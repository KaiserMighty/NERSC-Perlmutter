
#include <hip/hip_runtime.h>
#include <chrono>
#include <iomanip>
#include <iostream>
#include <math.h>

__global__
void add(int n, float *x, float *y)
{
    for (int i = 0; i < n; i++)
        y[i] = x[i] + y[i];
}

int main(void)
{
    std::cout << std::fixed << std::setprecision(5);

    int N = 1<<29;
    float *x, *y;
    hipMallocManaged(&x, N*sizeof(float));
    hipMallocManaged(&y, N*sizeof(float));

    for (int i = 0; i < N; i++)
    {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    std::chrono::time_point<std::chrono::high_resolution_clock> start_time = std::chrono::high_resolution_clock::now();

    add<<<1, 1>>>(N, x, y);

    std::chrono::time_point<std::chrono::high_resolution_clock> end_time = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed = end_time - start_time;
    std::cout << " Elapsed time is : " << elapsed.count() << " " << std::endl;

    float maxError = 0.0f;
    for (int i = 0; i < N; i++)
        maxError = fmax(maxError, fabs(y[i]-3.0f));
    std::cout << "Max error: " << maxError << std::endl;

    hipFree(x);
    hipFree(y);

    return 0;
}